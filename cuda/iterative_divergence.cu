
#include <hip/hip_runtime.h>
#include <iostream>
#include "stdio.h"
#define R 1 
#define C 32
#define H 1
#define numThrdx 32
#define numThrdy 1
#define numThrdz 1
 
#define ITERS 1000000000

#define g 64

__device__ uint get_smid(void) {
     uint ret;
     asm("mov.u32 %0, %smid;" : "=r"(ret) );
     return ret;
}

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;
    std::cout << "NBody.GPU" <<"\n" << "=========" <<"\n" <<"\n";

    std::cout << "CUDA version:   v" << CUDART_VERSION <<"\n";    
    //std::cout << "Thrust version: v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION <<"\n" <<"\n"; 

    int devCount;
    hipGetDeviceCount(&devCount);
    std::cout << "CUDA Devices: " <<"\n" <<"\n";

    for(int i = 0; i < devCount; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        std::cout << i << ": " << props.name << ": " << props.major << "." << props.minor <<"\n";
        std::cout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" <<"\n";
        std::cout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" <<"\n";
        std::cout << "  Constant memory: " << props.totalConstMem / kb << "kb" <<"\n";
        std::cout << "  Block registers: " << props.regsPerBlock <<"\n" <<"\n";

        std::cout << "  Warp size:         " << props.warpSize <<"\n";
        std::cout << "  Threads per block: " << props.maxThreadsPerBlock <<"\n";
        std::cout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", "<< props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2]<<" ]" <<"\n";
        std::cout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" <<"\n";
        std::cout <<"\n";
    }
}


__device__ void iteratively_divergent(int *a, int* w, int *k, int i) {

   __syncthreads();
   int gtid = gridDim.x*blockDim.x*gridDim.y*blockDim.y*threadIdx.z + (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y + blockDim.x*threadIdx.y + threadIdx.x;
   
   for(int i=0; i<32 ;i++)
      if(gtid <= i)
      {
         continue; 
      }      
      else
      {
     //   atomicAdd(&k[0],1);
        k[0]+=1;
        a[gtid] = k[0];
        w[gtid] = clock();
      }

}


__global__ void full_divergent(int *a, int* w, int *k) {
__syncthreads();

iteratively_divergent(a,w,k,0);
//store last value of k in a[0]
int stime = clock();
a[0] = 1;
int ftime = clock();
a[0] = ftime - stime;
}

__global__ void zero_divergent(int *a, int *k) {
int gtid = gridDim.x*blockDim.x*gridDim.y*blockDim.y*threadIdx.z + (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y + blockDim.x*threadIdx.y + threadIdx.x;

int j = 0;
while(j++ < ITERS);
   a[gtid] = (99 + get_smid());

}


int main() {

DisplayHeader();

int a[R][C][H], k[1], w[R][C][H];
int *dev_a, *dev_k, *dev_warp;

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipMalloc((void **) &dev_a, R*C*H*sizeof(int));
hipMalloc((void **) &dev_warp, R*C*H*sizeof(int));
hipMalloc((void **) &dev_k, sizeof(int));

// Fill Arrays
for (int k = 0; k < H; k++) {
   for (int i = 0; i < R; i++) {
      for (int j = 0; j < C; j++) {
    a[k][i][j] = 0;
    w[k][i][j] = 0;
    }
   }
}
k[0] = 0;

hipMemcpy(dev_a, a, R*C*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_warp, w, R*C*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(dev_k, k, sizeof(k), hipMemcpyHostToDevice);

// Kernel invocation 
dim3 threadsPerBlock(numThrdx,numThrdy,numThrdz); 
dim3 numBlocks( C/threadsPerBlock.x, R/threadsPerBlock.y, H/threadsPerBlock.z );

std::cout<<"numBlocks.x="<<numBlocks.x<<" numBlocks.y="<<numBlocks.y<<" numBlocks.z="<<numBlocks.z<<"\n";

hipEventRecord(start);
full_divergent<<<numBlocks, threadsPerBlock>>>(dev_a, dev_warp, dev_k);
hipEventRecord(stop);
hipEventSynchronize(stop);

hipMemcpy(a, dev_a, R*C*sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(w, dev_warp, R*C*sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(k, dev_k, sizeof(int), hipMemcpyDeviceToHost);

float elapsed_time = 0;
hipEventElapsedTime(&elapsed_time, start, stop);

std::cout<<"results:\n";
for (int k = 0; k < H; k++) {
   for (int i = 0; i < R; i++) {
	for (int j = 0; j < C; j++) {
            std::cout << a[k][i][j] <<" ";
    }
std::cout<<"\n";
   }
}

std::cout<<"clocks:\n";
for (int k = 0; k < H; k++) {
   for (int i = 0; i < R; i++) {
      for (int j = 0; j < C; j++) {
         std::cout << w[k][i][j] <<" ";
    }
std::cout<<"\n";
   }
}


std::cout<<"numBlocks.x="<<numBlocks.x<<" numBlocks.y="<<numBlocks.y<<" numBlocks.z="<<numBlocks.z<<"\n";
std::cout<<"Elapsed time = "<<elapsed_time<<" ms\n";

hipDeviceReset();
return 0;
}
